#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include "src/main.hxx"

using namespace std;




template <class G, class T>
void printRow(float t, const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  int repeat = 5; float tp = t - repeat*b.time;
  print(x); printf(" [%09.3f ms + %09.3f ms; %03d iters.] [%.4e err.] %s\n", tp, b.time, b.iterations, e, tec);
}

void runPagerankBatch(const string& data, int repeat, int skip, int batch) {
  using T = float;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  PagerankOptions<T> o = {repeat, Li, true};
  PagerankResult<T> a0, b0, c0, e0, f0;
  PagerankResult<T> b1, c1, d1, b2, c2, d2, b3, c3, d3;
  PagerankResult<T> b4, c4, d4, b5, c5, d5, b6, c6, d6;
  PagerankResult<T> e1, f1, g1, e2, f2, g2, e3, f3, g3;
  PagerankResult<T> e4, f4, g4, e5, f5, g5, e6, f6, g6;

  DiGraph<> xo;
  stringstream s(data);
  while (true) {
    // Skip some edges (to speed up execution)
    if (skip>0 && !readSnapTemporal(xo, s, skip)) break;
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    a0 = pagerankNvgraph(x, xt, init, o);
    auto r0 = a0.ranks;

    // Read edges for this batch.
    auto yo = copy(xo);
    if (!readSnapTemporal(yo, s, batch)) break;
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS:
    // Adjust ranks for insertions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find nvGraph-based pagerank.
    float tb0 = measureDuration([&]() { b0 = pagerankNvgraph(y, yt, init, o); });
    printRow(tb0, y, b0, b0, "I:pagerankNvgraph (static)");
    float tc0 = measureDuration([&]() { c0 = pagerankNvgraph(y, yt, &s0, o); });
    printRow(tc0, y, b0, c0, "I:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    float tb1 = measureDuration([&]() { b1 = pagerankMonolithicSeq(y, yt, init, o); });
    printRow(tb1, y, b0, b1, "I:pagerankMonolithicSeq (static)");
    float tc1 = measureDuration([&]() { c1 = pagerankMonolithicSeq(y, yt, &s0, o); });
    printRow(tc1, y, b0, c1, "I:pagerankMonolithicSeq (incremental)");
    float td1 = measureDuration([&]() { d1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, o); });
    printRow(td1, y, b0, d1, "I:pagerankMonolithicSeq (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    float tb2 = measureDuration([&]() { b2 = pagerankMonolithicOmp(y, yt, init, o); });
    printRow(tb2, y, b0, b2, "I:pagerankMonolithicOmp (static)");
    float tc2 = measureDuration([&]() { c2 = pagerankMonolithicOmp(y, yt, &s0, o); });
    printRow(tc2, y, b0, c2, "I:pagerankMonolithicOmp (incremental)");
    float td2 = measureDuration([&]() { d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, o); });
    printRow(td2, y, b0, d2, "I:pagerankMonolithicOmp (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    float tb3 = measureDuration([&]() { b3 = pagerankMonolithicCuda(y, yt, init, o); });
    printRow(tb3, y, b0, b3, "I:pagerankMonolithicCuda (static)");
    float tc3 = measureDuration([&]() { c3 = pagerankMonolithicCuda(y, yt, &s0, o); });
    printRow(tc3, y, b0, c3, "I:pagerankMonolithicCuda (incremental)");
    float td3 = measureDuration([&]() { d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, o); });
    printRow(td3, y, b0, d3, "I:pagerankMonolithicCuda (dynamic)");

    // Find sequential Levelwise pagerank.
    float tb4 = measureDuration([&]() { b4 = pagerankLevelwiseSeq(y, yt, init, o); });
    printRow(tb4, y, b0, b4, "I:pagerankLevelwiseSeq (static)");
    float tc4 = measureDuration([&]() { c4 = pagerankLevelwiseSeq(y, yt, &s0, o); });
    printRow(tc4, y, b0, c4, "I:pagerankLevelwiseSeq (incremental)");
    float td4 = measureDuration([&]() { d4 = pagerankLevelwiseSeqDynamic(x, xt, y, yt, &s0, o); });
    printRow(td4, y, b0, d4, "I:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    float tb5 = measureDuration([&]() { b5 = pagerankLevelwiseOmp(y, yt, init, o); });
    printRow(tb5, y, b0, b5, "I:pagerankLevelwiseOmp (static)");
    float tc5 = measureDuration([&]() { c5 = pagerankLevelwiseOmp(y, yt, &s0, o); });
    printRow(tc5, y, b0, c5, "I:pagerankLevelwiseOmp (incremental)");
    float td5 = measureDuration([&]() { d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, o); });
    printRow(td5, y, b0, d5, "I:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    float tb6 = measureDuration([&]() { b6 = pagerankLevelwiseCuda(y, yt, init, o); });
    printRow(tb6, y, b0, b6, "I:pagerankLevelwiseCuda (static)");
    float tc6 = measureDuration([&]() { c6 = pagerankLevelwiseCuda(y, yt, &s0, o); });
    printRow(tc6, y, b0, c6, "I:pagerankLevelwiseCuda (incremental)");
    float td6 = measureDuration([&]() { d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, o); });
    printRow(td6, y, b0, d6, "I:pagerankLevelwiseCuda (dynamic)");

    // DELETIONS:
    // Adjust ranks for deletions.
    auto s1 = b0.ranks;
    vector<T> r1(x.span());
    adjustRanks(r1, s1, ks, ksOld, 0.0f, float(Y)/(X+1), 1.0f/(X+1));

    // Find nvGraph-based pagerank.
    float te0 = measureDuration([&]() { e0 = pagerankNvgraph(x, xt, init, o); });
    printRow(te0, y, e0, e0, "D:pagerankNvgraph (static)");
    float tf0 = measureDuration([&]() { f0 = pagerankNvgraph(x, xt, &r1, o); });
    printRow(tf0, y, e0, f0, "D:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    float te1 = measureDuration([&]() { e1 = pagerankMonolithicSeq(x, xt, init, o); });
    printRow(te1, y, e0, e1, "D:pagerankMonolithicSeq (static)");
    float tf1 = measureDuration([&]() { f1 = pagerankMonolithicSeq(x, xt, &r1, o); });
    printRow(tf1, y, e0, f1, "D:pagerankMonolithicSeq (incremental)");
    float tg1 = measureDuration([&]() { g1 = pagerankMonolithicSeqDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg1, y, e0, g1, "D:pagerankMonolithicSeq (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    float te2 = measureDuration([&]() { e2 = pagerankMonolithicOmp(x, xt, init, o); });
    printRow(te2, y, e0, e2, "D:pagerankMonolithicOmp (static)");
    float tf2 = measureDuration([&]() { f2 = pagerankMonolithicOmp(x, xt, &r1, o); });
    printRow(tf2, y, e0, f2, "D:pagerankMonolithicOmp (incremental)");
    float tg2 = measureDuration([&]() { g2 = pagerankMonolithicOmpDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg2, y, e0, g2, "D:pagerankMonolithicOmp (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    float te3 = measureDuration([&]() { e3 = pagerankMonolithicCuda(x, xt, init, o); });
    printRow(te3, y, e0, e3, "D:pagerankMonolithicCuda (static)");
    float tf3 = measureDuration([&]() { f3 = pagerankMonolithicCuda(x, xt, &r1, o); });
    printRow(tf3, y, e0, f3, "D:pagerankMonolithicCuda (incremental)");
    float tg3 = measureDuration([&]() { g3 = pagerankMonolithicCudaDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg3, y, e0, g3, "D:pagerankMonolithicCuda (dynamic)");

    // Find sequential Levelwise pagerank.
    float te4 = measureDuration([&]() { e4 = pagerankLevelwiseSeq(x, xt, init, o); });
    printRow(te4, y, e0, e4, "D:pagerankLevelwiseSeq (static)");
    float tf4 = measureDuration([&]() { f4 = pagerankLevelwiseSeq(x, xt, &r1, o); });
    printRow(tf4, y, e0, f4, "D:pagerankLevelwiseSeq (incremental)");
    float tg4 = measureDuration([&]() { g4 = pagerankLevelwiseSeqDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg4, y, e0, g4, "D:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    float te5 = measureDuration([&]() { e5 = pagerankLevelwiseOmp(x, xt, init, o); });
    printRow(te5, y, e0, e5, "D:pagerankLevelwiseOmp (static)");
    float tf5 = measureDuration([&]() { f5 = pagerankLevelwiseOmp(x, xt, &r1, o); });
    printRow(tf5, y, e0, f5, "D:pagerankLevelwiseOmp (incremental)");
    float tg5 = measureDuration([&]() { g5 = pagerankLevelwiseOmpDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg5, y, e0, g5, "D:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    float te6 = measureDuration([&]() { e6 = pagerankLevelwiseCuda(x, xt, init, o); });
    printRow(te6, y, e0, e6, "D:pagerankLevelwiseCuda (static)");
    float tf6 = measureDuration([&]() { f6 = pagerankLevelwiseCuda(x, xt, &r1, o); });
    printRow(tf6, y, e0, f6, "D:pagerankLevelwiseCuda (incremental)");
    float tg6 = measureDuration([&]() { g6 = pagerankLevelwiseCudaDynamic(y, yt, x, xt, &r1, o); });
    printRow(tg6, y, e0, g6, "D:pagerankLevelwiseCuda (dynamic)");

    // New graph is now old.
    xo = move(yo);
  }
}


void runPagerank(const string& data, int repeat) {
  int M = countLines(data), steps = 10;
  printf("Temporal edges: %d\n", M);
  for (int batch=10, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, repeat, skip, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, repeat);
    printf("\n");
  return 0;
}

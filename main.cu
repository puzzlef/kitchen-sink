#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include "src/main.hxx"

using namespace std;




#define REPEAT 1

void runPagerankBatch(const string& data, bool show, int skip, int batch) {
  vector<float>  ranksAdj;
  vector<float> *initStatic  = nullptr;
  vector<float> *initDynamic = &ranksAdj;

  DiGraph<> x;
  stringstream s(data);
  while (true) {
    // Skip some edges (to speed up execution)
    if (!readSnapTemporal(x, s, skip)) break;
    loopDeadEnds(x);
    auto xt = transposeWithDegree(x);
    auto a1 = pagerankNvgraph(xt);
    auto ksOld    = vertices(x);
    auto ranksOld = move(a1.ranks);

    // Read edges for this batch.
    auto y = copy(x);
    if (!readSnapTemporal(y, s, batch)) break;
    loopDeadEnds(y);
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);

    // Adjust ranks using scaled-fill.
    ranksAdj.resize(y.span());
    adjustRanks(ranksAdj, ranksOld, ksOld, ks, 0.0f, float(ksOld.size())/ks.size(), 1.0f/ks.size());

    // Find static pagerank using nvGraph.
    auto a2 = pagerankNvgraph(yt, initStatic, {REPEAT});
    auto e2 = l1Norm(a2.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph [static]\n", a2.time, a2.iterations, e2);

    // Find dynamic pagerank using nvGraph.
    auto a3 = pagerankNvgraph(yt, initDynamic, {REPEAT});
    auto e3 = l1Norm(a3.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankNvgraph [dynamic]\n", a3.time, a3.iterations, e3);

    // Find static pagerank (monolithic).
    auto a4 = pagerankMonolithic(yt, initStatic, {REPEAT});
    auto e4 = l1Norm(a4.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithic [static]\n", a4.time, a4.iterations, e4);

    // Find dynamic pagerank (monolithic).
    auto a5 = pagerankMonolithic(yt, initDynamic, {REPEAT});
    auto e5 = l1Norm(a5.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithic [dynamic]\n", a5.time, a5.iterations, e5);

    // Find static levelwise pagerank.
    auto a6 = pagerankLevelwise(y, yt, initStatic, {REPEAT});
    auto e6 = l1Norm(a6.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankLevelwise [static]\n", a6.time, a6.iterations, e6);

    // Find dynamic levelwise pagerank.
    auto a7 = pagerankLevelwise(x, xt, y, yt, initDynamic, {REPEAT});
    auto e7 = l1Norm(a7.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankLevelwise [dynamic]\n", a7.time, a7.iterations, e7);

    // Find CUDA based static pagerank (monolithic).
    auto a8 = pagerankMonolithicCuda(yt, initStatic, {REPEAT});
    auto e8 = l1Norm(a8.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithicCuda [static]\n", a8.time, a8.iterations, e8);

    // Find CUDA based dynamic pagerank (monolithic).
    auto a9 = pagerankMonolithicCuda(yt, initDynamic, {REPEAT});
    auto e9 = l1Norm(a9.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankMonolithicCuda [dynamic]\n", a9.time, a9.iterations, e9);

    // Find CUDA based static levelwise pagerank.
    auto a10 = pagerankLevelwiseCuda(y, yt, initStatic, {REPEAT});
    auto e10 = l1Norm(a10.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankLevelwiseCuda [static]\n", a10.time, a10.iterations, e10);

    // Find CUDA based dynamic levelwise pagerank.
    auto a11 = pagerankLevelwiseCuda(x, xt, y, yt, initDynamic, {REPEAT});
    auto e11 = l1Norm(a11.ranks, a2.ranks);
    print(yt); printf(" [%09.3f ms; %03d iters.] [%.4e err.] pagerankLevelwiseCuda [dynamic]\n", a11.time, a11.iterations, e11);

    x = move(y);
  }
}


void runPagerank(const string& data, bool show) {
  int M = countLines(data), steps = 100;
  printf("Temporal edges: %d\n", M);
  for (int batch=100, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, show, skip, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  bool  show = argc > 2;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, show);
    printf("\n");
  return 0;
}

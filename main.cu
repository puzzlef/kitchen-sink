#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include "src/main.hxx"

using namespace std;




template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}

void runPagerankBatch(const string& data, int repeat, int skip, int batch) {
  vector<float> r0, s0;
  vector<float> *init = nullptr;
  PagerankOptions<float> o = {repeat};

  DiGraph<> xo;
  stringstream s(data);
  while (true) {
    // Skip some edges (to speed up execution)
    if (skip>0 && !readSnapTemporal(xo, s, skip)) break;
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankNvgraph(x, xt, init, o);
    auto r0 = move(a0.ranks);

    // Read edges for this batch.
    auto yo = copy(xo);
    if (!readSnapTemporal(yo, s, batch)) break;
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    s0.resize(y.span());

    // Adjust ranks.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(ksOld.size())/ks.size(), 1.0f/ks.size());

    // Find nvGraph-based pagerank.
    auto b0 = pagerankNvgraph(y, yt, init, o);
    printRow(y, b0, b0, "pagerankNvgraph (static)");
    auto c0 = pagerankNvgraph(y, yt, &s0, o);
    printRow(y, b0, c0, "pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    auto b1 = pagerankMonolithicSeq(y, yt, init, o);
    printRow(y, b0, b1, "pagerankMonolithicSeq (static)");
    auto c1 = pagerankMonolithicSeq(y, yt, &s0, o);
    printRow(y, b0, c1, "pagerankMonolithicSeq (incremental)");
    auto d1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d1, "pagerankMonolithicSeq (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    auto b2 = pagerankMonolithicOmp(y, yt, init, o);
    printRow(y, b0, b2, "pagerankMonolithicOmp (static)");
    auto c2 = pagerankMonolithicOmp(y, yt, &s0, o);
    printRow(y, b0, c2, "pagerankMonolithicOmp (incremental)");
    auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d2, "pagerankMonolithicOmp (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    auto b3 = pagerankMonolithicCuda(y, yt, init, o);
    printRow(y, b0, b3, "pagerankMonolithicCuda (static)");
    auto c3 = pagerankMonolithicCuda(y, yt, &s0, o);
    printRow(y, b0, c3, "pagerankMonolithicCuda (incremental)");
    auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d3, "pagerankMonolithicCuda (dynamic)");

    // Find sequential Levelwise pagerank.
    auto b4 = pagerankLevelwiseSeq(y, yt, init, o);
    printRow(y, b0, b4, "pagerankLevelwiseSeq (static)");
    auto c4 = pagerankLevelwiseSeq(y, yt, &s0, o);
    printRow(y, b0, c4, "pagerankLevelwiseSeq (incremental)");
    auto d4 = pagerankLevelwiseSeqDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d4, "pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, o);
    printRow(y, b0, b5, "pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, o);
    printRow(y, b0, c5, "pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d5, "pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    // auto b6 = pagerankLevelwiseCuda(y, yt, init, o);
    // printRow(y, b0, b6, "pagerankLevelwiseCuda (static)");
    // auto c6 = pagerankLevelwiseCuda(y, yt, &s0, o);
    // printRow(y, b0, c6, "pagerankLevelwiseCuda (incremental)");
    // auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, o);
    // printRow(y, b0, d6, "pagerankLevelwiseCuda (dynamic)");

    // New graph is now old.
    xo = move(yo);
  }
}


void runPagerank(const string& data, int repeat) {
  int M = countLines(data), steps = 100;
  printf("Temporal edges: %d\n", M);
  for (int batch=10, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, repeat, skip, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, repeat);
    printf("\n");
  return 0;
}

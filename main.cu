#include "hip/hip_runtime.h"
#include <cmath>
#include <string>
#include <vector>
#include <sstream>
#include <cstdio>
#include <iostream>
#include <utility>
#include "src/main.hxx"

using namespace std;




template <class G, class T>
void printRow(const G& x, const PagerankResult<T>& a, const PagerankResult<T>& b, const char *tec) {
  auto e = l1Norm(b.ranks, a.ranks);
  print(x); printf(" [%09.3f ms; %03d iters.] [%.4e err.] %s\n", b.time, b.iterations, e, tec);
}

void runPagerankBatch(const string& data, int repeat, int skip, int batch) {
  using T = float;
  using G = DiGraph<>;
  enum NormFunction { L0=0, L1=1, L2=2, Li=3 };
  vector<T> r0, s0, r1, s1;
  vector<T> *init = nullptr;
  PagerankOptions<T> o = {repeat, Li, true};

  DiGraph<> xo;
  stringstream s(data);
  while (true) {
    // Skip some edges (to speed up execution)
    if (skip>0 && !readSnapTemporal(xo, s, skip)) break;
    auto x  = selfLoop(xo, [&](int u) { return isDeadEnd(xo, u); });
    auto xt = transposeWithDegree(x);
    auto ksOld = vertices(x);
    auto a0 = pagerankNvgraph(x, xt, init, o);
    auto r0 = a0.ranks;

    // Read edges for this batch.
    auto yo = copy(xo);
    if (!readSnapTemporal(yo, s, batch)) break;
    auto y  = selfLoop(yo, [&](int u) { return isDeadEnd(yo, u); });
    auto yt = transposeWithDegree(y);
    auto ks = vertices(y);
    vector<T> s0(y.span());
    int X = ksOld.size();
    int Y = ks.size();

    // INSERTIONS:
    // Adjust ranks for insertions.
    adjustRanks(s0, r0, ksOld, ks, 0.0f, float(X)/(Y+1), 1.0f/(Y+1));

    // Find nvGraph-based pagerank.
    auto b0 = pagerankNvgraph(y, yt, init, o);
    printRow(y, b0, b0, "I:pagerankNvgraph (static)");
    auto c0 = pagerankNvgraph(y, yt, &s0, o);
    printRow(y, b0, c0, "I:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    auto b1 = pagerankMonolithicSeq(y, yt, init, o);
    printRow(y, b0, b1, "I:pagerankMonolithicSeq (static)");
    auto c1 = pagerankMonolithicSeq(y, yt, &s0, o);
    printRow(y, b0, c1, "I:pagerankMonolithicSeq (incremental)");
    auto d1 = pagerankMonolithicSeqDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d1, "I:pagerankMonolithicSeq (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    auto b2 = pagerankMonolithicOmp(y, yt, init, o);
    printRow(y, b0, b2, "I:pagerankMonolithicOmp (static)");
    auto c2 = pagerankMonolithicOmp(y, yt, &s0, o);
    printRow(y, b0, c2, "I:pagerankMonolithicOmp (incremental)");
    auto d2 = pagerankMonolithicOmpDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d2, "I:pagerankMonolithicOmp (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    auto b3 = pagerankMonolithicCuda(y, yt, init, o);
    printRow(y, b0, b3, "I:pagerankMonolithicCuda (static)");
    auto c3 = pagerankMonolithicCuda(y, yt, &s0, o);
    printRow(y, b0, c3, "I:pagerankMonolithicCuda (incremental)");
    auto d3 = pagerankMonolithicCudaDynamic(x, xt, y, yt, &s0, o);
    printRow(y, b0, d3, "I:pagerankMonolithicCuda (dynamic)");

    // Find sequential Levelwise pagerank.
    auto cs = components(y, yt);
    auto b  = blockgraph(y, cs);
    sortComponents(cs, b);
    PagerankData<G> D {move(cs), move(b)};
    auto b4 = pagerankLevelwiseSeq(y, yt, init, o, D);
    printRow(y, b0, b4, "I:pagerankLevelwiseSeq (static)");
    auto c4 = pagerankLevelwiseSeq(y, yt, &s0, o, D);
    printRow(y, b0, c4, "I:pagerankLevelwiseSeq (incremental)");
    auto d4 = pagerankLevelwiseSeqDynamic(x, xt, y, yt, &s0, o, D);
    printRow(y, b0, d4, "I:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto b5 = pagerankLevelwiseOmp(y, yt, init, o, D);
    printRow(y, b0, b5, "I:pagerankLevelwiseOmp (static)");
    auto c5 = pagerankLevelwiseOmp(y, yt, &s0, o, D);
    printRow(y, b0, c5, "I:pagerankLevelwiseOmp (incremental)");
    auto d5 = pagerankLevelwiseOmpDynamic(x, xt, y, yt, &s0, o, D);
    printRow(y, b0, d5, "I:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    auto b6 = pagerankLevelwiseCuda(y, yt, init, o, D);
    printRow(y, b0, b6, "I:pagerankLevelwiseCuda (static)");
    auto c6 = pagerankLevelwiseCuda(y, yt, &s0, o, D);
    printRow(y, b0, c6, "I:pagerankLevelwiseCuda (incremental)");
    auto d6 = pagerankLevelwiseCudaDynamic(x, xt, y, yt, &s0, o, D);
    printRow(y, b0, d6, "I:pagerankLevelwiseCuda (dynamic)");

    // DELETIONS:
    // Adjust ranks for deletions.
    auto s1 = b0.ranks;
    vector<T> r1(x.span());
    adjustRanks(r1, s1, ks, ksOld, 0.0f, float(Y)/(X+1), 1.0f/(X+1));

    // Find nvGraph-based pagerank.
    auto e0 = pagerankNvgraph(x, xt, init, o);
    printRow(y, e0, e0, "D:pagerankNvgraph (static)");
    auto f0 = pagerankNvgraph(x, xt, &r1, o);
    printRow(y, e0, f0, "D:pagerankNvgraph (incremental)");

    // Find sequential Monolithic pagerank.
    auto e1 = pagerankMonolithicSeq(x, xt, init, o);
    printRow(y, e0, e1, "D:pagerankMonolithicSeq (static)");
    auto f1 = pagerankMonolithicSeq(x, xt, &r1, o);
    printRow(y, e0, f1, "D:pagerankMonolithicSeq (incremental)");
    auto g1 = pagerankMonolithicSeqDynamic(y, yt, x, xt, &r1, o);
    printRow(y, e0, g1, "D:pagerankMonolithicSeq (dynamic)");

    // Find OpenMP-based Monolithic pagerank.
    auto e2 = pagerankMonolithicOmp(x, xt, init, o);
    printRow(y, e0, e2, "D:pagerankMonolithicOmp (static)");
    auto f2 = pagerankMonolithicOmp(x, xt, &r1, o);
    printRow(y, e0, f2, "D:pagerankMonolithicOmp (incremental)");
    auto g2 = pagerankMonolithicOmpDynamic(y, yt, x, xt, &r1, o);
    printRow(y, e0, g2, "D:pagerankMonolithicOmp (dynamic)");

    // Find CUDA-based Monolithic pagerank.
    auto e3 = pagerankMonolithicCuda(x, xt, init, o);
    printRow(y, e0, e3, "D:pagerankMonolithicCuda (static)");
    auto f3 = pagerankMonolithicCuda(x, xt, &r1, o);
    printRow(y, e0, f3, "D:pagerankMonolithicCuda (incremental)");
    auto g3 = pagerankMonolithicCudaDynamic(y, yt, x, xt, &r1, o);
    printRow(y, e0, g3, "D:pagerankMonolithicCuda (dynamic)");

    // Find sequential Levelwise pagerank.
    auto ds = components(x, xt);
    auto c  = blockgraph(x, ds);
    sortComponents(ds, c);
    PagerankData<G> E {move(ds), move(c)};
    auto e4 = pagerankLevelwiseSeq(x, xt, init, o, E);
    printRow(y, e0, e4, "D:pagerankLevelwiseSeq (static)");
    auto f4 = pagerankLevelwiseSeq(x, xt, &r1, o, E);
    printRow(y, e0, f4, "D:pagerankLevelwiseSeq (incremental)");
    auto g4 = pagerankLevelwiseSeqDynamic(y, yt, x, xt, &r1, o, E);
    printRow(y, e0, g4, "D:pagerankLevelwiseSeq (dynamic)");

    // Find OpenMP-based Levelwise pagerank.
    auto e5 = pagerankLevelwiseOmp(x, xt, init, o, E);
    printRow(y, e0, e5, "D:pagerankLevelwiseOmp (static)");
    auto f5 = pagerankLevelwiseOmp(x, xt, &r1, o, E);
    printRow(y, e0, f5, "D:pagerankLevelwiseOmp (incremental)");
    auto g5 = pagerankLevelwiseOmpDynamic(y, yt, x, xt, &r1, o, E);
    printRow(y, e0, g5, "D:pagerankLevelwiseOmp (dynamic)");

    // Find CUDA-based Levelwise pagerank.
    auto e6 = pagerankLevelwiseCuda(x, xt, init, o, E);
    printRow(y, e0, e6, "D:pagerankLevelwiseCuda (static)");
    auto f6 = pagerankLevelwiseCuda(x, xt, &r1, o, E);
    printRow(y, e0, f6, "D:pagerankLevelwiseCuda (incremental)");
    auto g6 = pagerankLevelwiseCudaDynamic(y, yt, x, xt, &r1, o, E);
    printRow(y, e0, g6, "D:pagerankLevelwiseCuda (dynamic)");

    // New graph is now old.
    xo = move(yo);
  }
}


void runPagerank(const string& data, int repeat) {
  int M = countLines(data), steps = 10;
  printf("Temporal edges: %d\n", M);
  for (int batch=10, i=0; batch<M; batch*=i&1? 2:5, i++) {
    int skip = max(M/steps - batch, 0);
    printf("\n# Batch size %.0e\n", (double) batch);
    runPagerankBatch(data, repeat, skip, batch);
  }
}


int main(int argc, char **argv) {
  char *file = argv[1];
  int repeat = argc>2? stoi(argv[2]) : 5;
  printf("Using graph %s ...\n", file);
  string d = readFile(file);
  runPagerank(d, repeat);
    printf("\n");
  return 0;
}
